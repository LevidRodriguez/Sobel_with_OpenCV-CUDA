#include "hip/hip_runtime.h"
#include <thread>
#include <chrono>
#include <time.h>
#include <iostream>
#include <math.h>
// #include "imageLoader.cpp"
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/stitching.hpp>
#include <opencv2/core/utility.hpp>

#define GridSize 20.0 
void sobelFilterCPU(cv::Mat srcImg, cv::Mat dstImg, const unsigned int width, const unsigned int height);
void sobelFilterOpenCV(cv::Mat srcImg, cv::Mat dstImg);

__global__ void sobelFilterGPU(unsigned char* srcImg, unsigned char* dstImg, const unsigned int width, const unsigned int height){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    float dx, dy;
    if( x > 0 && y > 0 && x < width-1 && y < height-1) {
        dx = (-1* srcImg[(y-1)*width + (x-1)]) + (-2*srcImg[y*width+(x-1)]) + (-1*srcImg[(y+1)*width+(x-1)]) +
             (    srcImg[(y-1)*width + (x+1)]) + ( 2*srcImg[y*width+(x+1)]) + (   srcImg[(y+1)*width+(x+1)]);
             
        dy = (    srcImg[(y-1)*width + (x-1)]) + ( 2*srcImg[(y-1)*width+x]) + (   srcImg[(y-1)*width+(x+1)]) +
             (-1* srcImg[(y+1)*width + (x-1)]) + (-2*srcImg[(y+1)*width+x]) + (-1*srcImg[(y+1)*width+(x+1)]);
        
        dstImg[y*width + x] = sqrt( (dx*dx) + (dy*dy) );
    }
}

int main(int argc, char * argv[]){
    if(argc != 2){
        std::cout << argv[0] << "Invalid number of command line arguments. Exiting program" << std::endl;
        std::cout << "Usage: " << argv[0] << " [image.png]"<< std::endl;
        return 1;
    }
    // Verifica las versiones de GPU, CUDA y OpenCV.
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    int cores = devProp.multiProcessorCount;
    switch (devProp.major){
	case 2: // Fermi
		if (devProp.minor == 1) cores *= 48;
		else cores *= 32; break;
	case 3: // Kepler
		cores *= 192; break;
	case 5: // Maxwell
		cores *= 128; break;
	case 6: // Pascal
		if (devProp.minor == 1) cores *= 128;
		else if (devProp.minor == 0) cores *= 64;
		break;
    }
    time_t rawTime;time(&rawTime);
    struct tm* curTime = localtime(&rawTime);
    char timeBuffer[80] = "";
    strftime(timeBuffer, 80, "edge map benchmarks (%c)\n", curTime);
    std::cout << timeBuffer << std::endl;
    std::cout << "GPGPU: " << devProp.name << ", CUDA "<< devProp.major << "."<< devProp.minor <<", "<< devProp.totalGlobalMem / 1048576 << 
                " Mbytes global memory, "<< cores << " CUDA cores\n" <<std::endl;
    std::cout << "OpenCV Version: " << CV_VERSION << std::endl;
    
    // Cargar imagen y la transforma a escala de grises
    cv::Mat srcImg = cv::imread(argv[1]); 
    cv::cvtColor(srcImg, srcImg, cv::COLOR_RGB2GRAY);
    cv::Mat sobel_cpu = cv::Mat::zeros(srcImg.size(),srcImg.type());
    cv::Mat sobel_opencv = cv::Mat::zeros(srcImg.size(), srcImg.type());
    
    unsigned char *gpu_orig, *gpu_sobel;
    auto c = std::chrono::system_clock::now();
    /******************************************---START CPU---****************************************************/
    sobelFilterCPU(srcImg, sobel_cpu, srcImg.cols, srcImg.rows);
    std::chrono::duration<double> time_cpu = std::chrono::system_clock::now() - c;    
    /******************************************---END CPU---******************************************************/
    
    /******************************************---START OPENCV---****************************************************/
    c = std::chrono::system_clock::now();
    sobelFilterOpenCV(srcImg, sobel_opencv);
    std::chrono::duration<double> time_opencv = std::chrono::system_clock::now() - c;    
    /******************************************---END OPENCV---******************************************************/

    /******************************************---SETUP GPU---****************************************************/
    // Asignar memoria para las imágenes en memoria GPU.
    hipMalloc( (void**)&gpu_orig, (srcImg.cols * srcImg.rows));
    hipMalloc( (void**)&gpu_sobel, (srcImg.cols * srcImg.rows));
    // Transfiera del host al device y configura la matriz resultante a 0s
    hipMemcpy(gpu_orig, srcImg.data, (srcImg.cols*srcImg.rows), hipMemcpyHostToDevice);
    hipMemset(gpu_sobel, 0, (srcImg.cols*srcImg.rows));
    // configura los dim3 para que gpu los use como argumentos, hilos por bloque y número de bloques
    dim3 threadsPerBlock(GridSize, GridSize, 1);
    dim3 numBlocks(ceil(srcImg.cols/GridSize), ceil(srcImg.rows/GridSize), 1);
    hipStream_t stream;
    hipStreamCreate(&stream);

    /******************************************---START GPU---****************************************************/
    // Ejecutar el filtro sobel utilizando la GPU.
    c = std::chrono::system_clock::now();
    sobelFilterGPU<<< numBlocks, threadsPerBlock, 0, stream >>>(gpu_orig, gpu_sobel, srcImg.cols, srcImg.rows);
    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    // if error, output error
    if ( cudaerror != hipSuccess ) 
        std::cout <<  "Cuda failed to synchronize: " << hipGetErrorName( cudaerror ) <<std::endl;
    std::chrono::duration<double> time_gpu = std::chrono::system_clock::now() - c;
    /******************************************---END GPU---****************************************************/
    // Copia los datos al CPU desde la GPU, del device al host
    hipMemcpy(srcImg. data, gpu_sobel, (srcImg.cols*srcImg.rows), hipMemcpyDeviceToHost);
    /** Tiempos de ejecución de cada método de filtrado por sobel **/
    std::cout << "\nProcessing "<< argv[1] << ": "<<srcImg.rows<<" rows x "<<srcImg.cols << " columns" << std::endl;
    std::cout << "CPU execution time   = " << 1000*time_cpu.count() <<" msec"<<std::endl;
    std::cout << "OPENCV execution time   = " << 1000*time_opencv.count() <<" msec"<<std::endl;
    std::cout << "CUDA execution time   = " << 1000*time_gpu.count() <<" msec"<<std::endl;
    // Save results
    cv::imwrite("outImgCPU.png",sobel_cpu);    
    cv::imwrite("outImgOpenCV.png",sobel_opencv);
    cv::imwrite("outImgGPU.png",srcImg);
    hipStreamDestroy(stream);    
    hipFree(gpu_orig); hipFree(gpu_sobel);

    return 0;
}

void sobelFilterCPU(cv::Mat srcImg, cv::Mat dstImg, const unsigned int width, const unsigned int height){
    // cv::cvtColor(orig, cpu, cv::COLOR_RGB2GRAY);
    for(int y = 1; y < srcImg.rows-1; y++) {
        for(int x = 1; x < srcImg.cols-1; x++) {
            int dx = (-1*srcImg.data[(y-1)*width + (x-1)]) + (-2*srcImg.data[y*width+(x-1)]) + (-1*srcImg.data[(y+1)*width+(x-1)]) +
            (srcImg.data[(y-1)*width + (x+1)]) + (2*srcImg.data[y*width+(x+1)]) + (srcImg.data[(y+1)*width+(x+1)]);
            int dy = (srcImg.data[(y-1)*width + (x-1)]) + (2*srcImg.data[(y-1)*width+x]) + (srcImg.data[(y-1)*width+(x+1)]) +
            (-1*srcImg.data[(y+1)*width + (x-1)]) + (-2*srcImg.data[(y+1)*width+x]) + (-1*srcImg.data[(y+1)*width+(x+1)]);
            // int sum = abs(dx) + abs(dy);
            int sum = sqrt((dx*dx)+(dy*dy));
            // sum = sum>255?255:sum;
            // cpu[y*width + x] = sqrt((dx*dx)+(dy*dy));
            dstImg.at<uchar>(y,x) = sum;
        }
    }
}

void sobelFilterOpenCV(cv::Mat srcImg, cv::Mat dstImg){
    cv::Mat grad_x, grad_y, abs_grad_x, abs_grad_y;
    // Gradiente X
    cv::Sobel(srcImg, grad_x, CV_16S, 1, 0, 3, 1, 0, cv::BORDER_DEFAULT);
    cv::convertScaleAbs(grad_x, abs_grad_x);
    // Gradiente Y
    cv::Sobel(srcImg, grad_y, CV_16S, 0, 1, 3, 1, 0, cv::BORDER_DEFAULT);
    cv::convertScaleAbs(grad_y, abs_grad_y);

    addWeighted( abs_grad_x, 0.5, abs_grad_y, 0.5, 0, dstImg );
}