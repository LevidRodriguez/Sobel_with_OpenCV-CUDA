#include "hip/hip_runtime.h"
#include <thread>
#include <chrono>
#include <time.h>
#include <iostream>
#include <math.h>
// #include "imageLoader.cpp"
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/stitching.hpp>
#include <opencv2/core/utility.hpp>

#define GridSize 20.0 
void sobel_cpu(unsigned char* orig, unsigned char* cpu, const unsigned int width, const unsigned int height);

__global__ void sobelFilterGPU(unsigned char* orig, unsigned char* cpu, const unsigned int width, const unsigned int height){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    float dx, dy;
    if( x > 0 && y > 0 && x < width-1 && y < height-1) {
        dx = (-1* orig[(y-1)*width + (x-1)]) + (-2*orig[y*width+(x-1)]) + (-1*orig[(y+1)*width+(x-1)]) +
             (    orig[(y-1)*width + (x+1)]) + ( 2*orig[y*width+(x+1)]) + (   orig[(y+1)*width+(x+1)]);
             
        dy = (    orig[(y-1)*width + (x-1)]) + ( 2*orig[(y-1)*width+x]) + (   orig[(y-1)*width+(x+1)]) +
             (-1* orig[(y+1)*width + (x-1)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+1)]);
        
        cpu[y*width + x] = sqrt( (dx*dx) + (dy*dy) );
    }
}

int main(int argc, char * argv[]){
    if(argc != 2){
        std::cout << argv[0] << "Invalid number of command line arguments. Exiting program" << std::endl;
        std::cout << "Usage: " << argv[0] << " [image.png]"<< std::endl;
        return 1;
    }
    // Verify GPU, CUDA and OpenCV version
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    int cores = devProp.multiProcessorCount;
    switch (devProp.major){
	case 2: // Fermi
		if (devProp.minor == 1) cores *= 48;
		else cores *= 32; break;
	case 3: // Kepler
		cores *= 192; break;
	case 5: // Maxwell
		cores *= 128; break;
	case 6: // Pascal
		if (devProp.minor == 1) cores *= 128;
		else if (devProp.minor == 0) cores *= 64;
		break;
    }
    time_t rawTime;time(&rawTime);
    struct tm* curTime = localtime(&rawTime);
    char timeBuffer[80] = "";
    strftime(timeBuffer, 80, "edge map benchmarks (%c)\n", curTime);
    std::cout << timeBuffer << std::endl;
    std::cout << "GPGPU: " << devProp.name << ", CUDA "<< devProp.major << "."<< devProp.minor <<", "<< devProp.totalGlobalMem / 1048576 << 
                " Mbytes global memory, "<< cores << " CUDA cores\n" <<std::endl;
    std::cout << "OpenCV Version: " << CV_VERSION << std::endl;
    // Load Image
    cv::Mat origImg = cv::imread(argv[1]);
    cv::cvtColor(origImg, origImg, cv::COLOR_RGB2GRAY);
    unsigned char *gpu_orig, *gpu_sobel, *cpu_sobel;
    auto c = std::chrono::system_clock::now();
    sobel_cpu(origImg.data, cpu_sobel, origImg.cols, origImg.cols);
    std::chrono::duration<double> time_cpu = std::chrono::system_clock::now() - c;    

    // Allocate memory for the images in GPU memory 
    hipMalloc( (void**)&gpu_orig, (origImg.cols * origImg.rows));
    hipMalloc( (void**)&gpu_sobel, (origImg.cols * origImg.rows));
    // Transfiera del host al device y configura la matriz resultante a 0s
    hipMemcpy(gpu_orig, origImg.data, (origImg.cols*origImg.rows), hipMemcpyHostToDevice);
    hipMemset(gpu_sobel, 0, (origImg.cols*origImg.rows));
    // configura los dim3 para que gpu los use como argumentos, hilos por bloque y número de bloques
    dim3 threadsPerBlock(GridSize, GridSize, 1);
    dim3 numBlocks(ceil(origImg.cols/GridSize), ceil(origImg.rows/GridSize), 1);
    
    // Ejecutar el filtro sobel utilizando la CPU.
    c = std::chrono::system_clock::now();
    sobelFilterGPU<<<numBlocks, threadsPerBlock>>>(gpu_orig, gpu_sobel, origImg.cols, origImg.rows);
    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    // if error, output error
    if ( cudaerror != hipSuccess ) 
        std::cout <<  "Cuda failed to synchronize: " << hipGetErrorName( cudaerror ) <<std::endl;
    
    std::chrono::duration<double> time_gpu = std::chrono::system_clock::now() - c;
    // Copia los datos al CPU desde la GPU, del device al host
    hipMemcpy(origImg. data, gpu_sobel, (origImg.cols*origImg.rows), hipMemcpyDeviceToHost);

    /** Output runtimes of each method of sobel filtering **/
    std::cout << "\nProcessing "<< argv[1] << ": "<<origImg.rows<<" rows x "<<origImg.cols << " columns" << std::endl;
    std::cout << "CPU execution time   = " << 1000*time_cpu.count() <<" msec"<<std::endl;
    std::cout << "CUDA execution time   = " << 1000*time_gpu.count() <<" msec"<<std::endl;

    // Save results
    cv::imwrite("outImg.png",origImg);    
    hipFree(gpu_orig); hipFree(gpu_sobel);

    return 0;
}

void sobel_cpu(unsigned char* orig, unsigned char* cpu,const unsigned int width, const unsigned int height) {
    for(int y = 1; y < height-1; y++) {
        for(int x = 1; x < width-1; x++) {
            int dx = (-1*orig[(y-1)*width + (x-1)]) + (-2*orig[y*width+(x-1)]) + (-1*orig[(y+1)*width+(x-1)]) +
                     (orig[(y-1)*width + (x+1)]) + (2*orig[y*width+(x+1)]) + (orig[(y+1)*width+(x+1)]);
            int dy = (orig[(y-1)*width + (x-1)]) + (2*orig[(y-1)*width+x]) + (orig[(y-1)*width+(x+1)]) +
                (-1*orig[(y+1)*width + (x-1)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+1)]);
                cpu[y*width + x] = sqrt((dx*dx)+(dy*dy));
            }
        }
    }