#include <thread>
#include <chrono>
#include <time.h>
#include <iostream>
#include <math.h>
#include "imageLoader.cpp"
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/stitching.hpp>
#include <opencv2/core/utility.hpp>


#define GRIDVAL 20.0 

int main(int argc, char * argv[]){
    if(argc != 2){
        std::cout << argv[0] << "Invalid number of command line arguments. Exiting program" << std::endl;
        std::cout << "Usage: " << argv[0] << " [image.png]"<< std::endl;
        return 1;
    }
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    int cores = devProp.multiProcessorCount;
    switch (devProp.major){
	case 2: // Fermi
		if (devProp.minor == 1) cores *= 48;
		else cores *= 32; break;
	case 3: // Kepler
		cores *= 192; break;
	case 5: // Maxwell
		cores *= 128; break;
	case 6: // Pascal
		if (devProp.minor == 1) cores *= 128;
		else if (devProp.minor == 0) cores *= 64;
		break;
    }
    time_t rawTime;time(&rawTime);
    struct tm* curTime = localtime(&rawTime);
    char timeBuffer[80] = "";
    strftime(timeBuffer, 80, "edge map benchmarks (%c)\n", curTime);
    std::cout << timeBuffer << std::endl;
    std::cout << "GPGPU: " << devProp.name << ", CUDA "<< devProp.major << "."<< devProp.minor <<", "<< devProp.totalGlobalMem / 1048576 << 
                " Mbytes global memory, "<< cores << " CUDA cores\n" <<std::endl;
    std::cout << "OpenCV Version: " << CV_VERSION << std::endl;
    return 0;
}